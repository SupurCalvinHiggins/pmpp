
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>

// A is n x n
// B is n x n
// C is n x n

#define TILE_WIDTH 32
#define RM(row, col, width) ((row) * (width) + (col))

__global__ void ker_matmul(const float *A, const float *B, float *C,
                           int width) {

  __shared__ float At[TILE_WIDTH][TILE_WIDTH];
  __shared__ float Bt[TILE_WIDTH][TILE_WIDTH];

  float acc[TILE_WIDTH];
  for (int r = 0; r < TILE_WIDTH; ++r) {
    acc[r] = 0.0;
  }

  const int tile_col = blockIdx.x;
  const int tile_row = blockIdx.y;
  const int elem_col = threadIdx.x;

  for (int p = 0; p < width / TILE_WIDTH; ++p) {
    // Copy data into At and Bt.
    // At is the tile at (row, p)
    // Bt is the tile at (p, col)
    for (int elem_row = 0; elem_row < TILE_WIDTH; ++elem_row) {
      int idx = RM(tile_row * TILE_WIDTH + elem_row, p * TILE_WIDTH + elem_col,
                   width);
      At[elem_row][elem_col] = A[idx];
      Bt[elem_row][elem_col] = B[RM(p * TILE_WIDTH + elem_row,
                                    tile_col * TILE_WIDTH + elem_col, width)];
    }
    __syncthreads();

    // Accumulate into acc.
    for (int elem_row = 0; elem_row < TILE_WIDTH; ++elem_row) {
      for (int i = 0; i < TILE_WIDTH; ++i) {
        acc[elem_row] += At[elem_row][i] * Bt[i][elem_col];
      }
    }
    __syncthreads();
  }

  for (int elem_row = 0; elem_row < TILE_WIDTH; ++elem_row) {
    C[RM(tile_row * TILE_WIDTH + elem_row, tile_col * TILE_WIDTH + elem_col,
         width)] = acc[elem_row];
  }
}

void matmul(const float *A, const float *B, float *C, int width) {
  float *A_d, *B_d, *C_d;

  hipMalloc(&A_d, width * width * sizeof(float));
  hipMalloc(&B_d, width * width * sizeof(float));
  hipMalloc(&C_d, width * width * sizeof(float));

  hipMemcpy(A_d, A, width * width * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(B_d, B, width * width * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(C_d, C, width * width * sizeof(float), hipMemcpyHostToDevice);

  const dim3 gridDim(width / TILE_WIDTH, width / TILE_WIDTH, 1);
  const dim3 blockDim(TILE_WIDTH, 1, 1);
  ker_matmul<<<gridDim, blockDim>>>(A_d, B_d, C_d, width);

  hipMemcpy(C, C_d, width * width * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);
}

int main(int argc, char *argv[]) {
  const int width = 8 * TILE_WIDTH;

  float A[width][width];
  float B[width][width];
  float C[width][width];

  for (int i = 0; i < width; ++i) {
    for (int j = 0; j < width; ++j) {
      A[i][j] = (2.0 * i * width - 3.0 * j) / (width * width);
      B[i][j] = (-j * width + 5.0 * i) / (width * width);
      C[i][j] = 0.0;
    }
  }

  matmul((const float *)A, (const float *)B, (float *)C, width);

  for (int i = 0; i < width; ++i) {
    for (int j = 0; j < width; ++j) {
      float acc = 0.0;
      for (int k = 0; k < width; ++k) {
        acc += A[i][k] * B[k][j];
      }
      // printf("C[%d][%d] == %f == %f\n", i, j, C[i][j], acc);
      assert(abs(C[i][j] - acc) < 0.001);
    }
  }
  printf("OK\n");
}
